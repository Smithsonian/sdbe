/*
 * GPU kernel code for the following pipeline components:
 *   VDIF interpreter
 *   B-engine depacketizer
 *   Pre-preprocessor
 */

#include <getopt.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <hipfft/hipfft.h>

// VDIF constants
#define VDIF_BYTE_SIZE 1056 // VDIF frame size in bytes
#define VDIF_BYTE_SIZE_HEADER 32 // VDIF header size in bytes
#define VDIF_BYTE_SIZE_DATA 1024 // VDIF data size in bytes
#define VDIF_INT_SIZE (1056/4) // VDIF frame size in int
#define VDIF_INT_SIZE_HEADER (32/4) // VDIF header size in int
#define VDIF_INT_SIZE_DATA (1024/4) // VDIF data size in int
#define VDIF_BIT_DEPTH 2 // bits-per-sample

// GPU Controls
#ifndef THREADS_PER_BLOCK_X
	#define THREADS_PER_BLOCK_X 32 //8 // set specifically so that each thread reads one int32_t in header
#endif
#ifndef THREADS_PER_BLOCK_Y
	#define THREADS_PER_BLOCK_Y 32 //4 // set so there is one warp per block, not necessarily optimal
#endif
//~ #define BLOCKS_PER_GRID 128 // arbitrary power of 2
//~ #define GPUCTRL_SHARED_RAW_VDIF

// Data structure
#define BENG_CHANNELS_ 16384
#define BENG_CHANNELS (BENG_CHANNELS_+1) // number of channels PLUS added sample-rate/2 component for the complex-to-real inverse transform
#define BENG_SNAPSHOTS 128
#define BENG_BUFFER_IN_COUNTS 4 // we will buffer 32 B-engine frames
#define BENG_BUFFER_INDEX_MASK (BENG_BUFFER_IN_COUNTS-1) // mask used to convert B-engine counter to index into buffer
#define SWARM_N_FIDS 8
#define SWARM_XENG_PARALLEL_CHAN 8
#define BENG_FRAMES_OUT_CONSECUTIVE_SNAPSHOTS // if defined, B-engine frames are stored such that the same spectral channel for consecutive snapshots are adjacent in memory

// VDIF packed B-engine packet
#define BENG_VDIF_HDR_0_OFFSET_INT 4 // b1 b2 b3 b4
#define BENG_VDIF_HDR_1_OFFSET_INT 5 //  c  0  f b0
#define BENG_VDIF_CHANNELS_PER_INT 4 // a-d in a single int32_t, and e-h in a single int32_t
#define BENG_VDIF_INT_PER_SNAPSHOT (SWARM_XENG_PARALLEL_CHAN/BENG_VDIF_CHANNELS_PER_INT)
#define BENG_PACKETS_PER_FRAME (BENG_CHANNELS_/SWARM_XENG_PARALLEL_CHAN)
#define BENG_FRAME_COMPLETION_COMPLETE_ON_GPU (BENG_PACKETS_PER_FRAME*blockDim.x) // value of completion counter when B-engine frame complete, multiplication by THREADS_PER_BLOCK_x required since all x-threads increment counter
#define BENG_FRAME_COMPLETION_COMPLETE_ON_CPU (BENG_PACKETS_PER_FRAME*num_x_threads) // value of completion counter when B-engine frame complete, multiplication by THREADS_PER_BLOCK_x required since all x-threads increment counter
#define BENG_VDIF_SAMPLE_VALUE_OFFSET 2.0f

// Debugging
//#define DEBUG
//~ #define DEBUG_GPU
//~ #define DEBUG_GPU_CONDITION (blockIdx.x == 0 && threadIdx.x == 7 && threadIdx.y == 3)
//~ #define DEBUG_SINGLE_FRAME
#define DEBUG_SINGLE_FRAME_CID 128
#define DEBUG_SINGLE_FRAME_FID 6
#define DEBUG_SINGLE_FRAME_BCOUNT 376264

/*
 *  Forward declarations
 */
// GPU kernels
__global__ void vdif_to_beng(
	int32_t *vdif_frames, 
	int32_t *fid_out, 
	int32_t *cid_out, 
	int32_t *bcount_out, 
	hipfftComplex *beng_data_out_0,
	hipfftComplex *beng_data_out_1, 
	int32_t *beng_frame_completion,
	int32_t num_vdif_frames, 
	int32_t bcount_offset,
	int blocks_per_grid);
// host utilities
inline void error_check(const char *f, const int l);
inline void error_check(hipError_t err, const char *f, const int l);

/*
 * Data handling inlines.
 */
// Read B-engine C-stamp from VDIF header
__host__ __device__ inline int32_t get_cid_from_vdif(const int32_t *vdif_start)
{
	return (*(vdif_start + BENG_VDIF_HDR_1_OFFSET_INT) & 0x000000FF);
}
// Read B-engine F-stamp from VDIF header
__host__ __device__ inline int32_t get_fid_from_vdif(const int32_t *vdif_start)
{
	return (*(vdif_start + BENG_VDIF_HDR_1_OFFSET_INT) & 0x00FF0000)>>16;
}
// Read B-engine B-counter from VDIF header
__host__ __device__ inline int32_t get_bcount_from_vdif(const int32_t *vdif_start)
{
	return ((*(vdif_start + BENG_VDIF_HDR_1_OFFSET_INT)&0xFF000000)>>24) + ((*(vdif_start + BENG_VDIF_HDR_0_OFFSET_INT)&0x00FFFFFF)<<8);
}
// Read complex sample pair and shift input data accordingly inplace.
__host__ __device__ inline hipfftComplex read_complex_sample(int32_t *samples_int)
{
	float sample_imag, sample_real;
	
	#ifdef __CUDA_ARCH__
		sample_imag = __int2float_rd(*samples_int & 0x03) - BENG_VDIF_SAMPLE_VALUE_OFFSET;
	#else
		sample_imag = (float)(*samples_int & 0x03) - BENG_VDIF_SAMPLE_VALUE_OFFSET;
	#endif
	*samples_int = (*samples_int) >> VDIF_BIT_DEPTH;
	#ifdef __CUDA_ARCH__
		sample_real = __int2float_rd(*samples_int & 0x03) - BENG_VDIF_SAMPLE_VALUE_OFFSET;
	#else
		sample_real = (float)(*samples_int & 0x03) - BENG_VDIF_SAMPLE_VALUE_OFFSET;
	#endif
	*samples_int = (*samples_int) >> VDIF_BIT_DEPTH;
	return make_hipFloatComplex(sample_real, sample_imag);
}

int main(int argc, char **argv)
{
	#ifdef DEBUG
	printf("reader:DEBUG:Start\n");
	#endif
	
	// misc
	bool verbose_output = 0, logging = 0, input_is_batch = 0;
	int blocks_per_grid = 128;
	char filename_log[0x100] = "\0";
	FILE *fh_log = NULL;
	int repeats = 1, ir;
	int ii,ij,ik,il;
	time_t wall_clock;
	int32_t tmp_vdif[VDIF_INT_SIZE];
	int32_t tmp_bcount_prev,tmp_bcount_curr;
	int num_x_threads = THREADS_PER_BLOCK_X, num_y_threads = THREADS_PER_BLOCK_Y;
	
	// input (host)
	FILE *fh = NULL;
	char filename_input[0x100] = "\0";
	int32_t num_vdif_frames = 0;
	int32_t *vdif_buf = NULL;
	int32_t beng_frame_offset = -1;
	int32_t bcount_offset = 0;
	
	// input (device)
	int32_t *gpu_vdif_buf = NULL;
	
	// output (host)
	int32_t *fid;
	int32_t *cid;
	int32_t *bcount;
	hipfftComplex *beng_data_0,*beng_data_1;
	FILE *fh_data = NULL;
	char filename_data[0x100] = "\0";
	bool data_to_file = 0; 
	
	// output (device)
	int32_t *gpu_fid;
	int32_t *gpu_cid;
	int32_t *gpu_bcount;
	hipfftComplex *gpu_beng_data_0,*gpu_beng_data_1;
	
	// iFFT module (host)
	hipfftHandle ifft_plan;
	int ifft_rank = 1;
	int ifft_size = BENG_CHANNELS_*2;
	int ifft_inembed[1] = {ifft_size}; 
	#ifdef BENG_FRAMES_OUT_CONSECUTIVE_SNAPSHOTS
		int ifft_istride = BENG_SNAPSHOTS*BENG_BUFFER_IN_COUNTS;
		int ifft_idist = 1;
	#else
		int ifft_istride = 1;
		int ifft_idist = BENG_CHANNELS;
	#endif
	int ifft_onembed[1] = {ifft_size};
	int ifft_ostride = 1;
	int ifft_odist = BENG_CHANNELS_*2;
	hipfftReal *time_series_0,*time_series_1;
	hipfftResult cures = HIPFFT_SUCCESS;
	
	// iFFT module (device)
	hipfftReal *gpu_time_series_0,*gpu_time_series_1;
	
	// control (host)
	int32_t *beng_frame_completion; // counts number of packets received per b-count value
	
	// control (device)
	int32_t *gpu_beng_frame_completion; // counts number of packets received per b-count value
	
	// for CUDA error checking
	hipError_t err;
	
	// for CUDA timing
	struct timespec t0,t1;
	hipEvent_t start, stop;
	float time_spent;
	
	#ifdef DEBUG
	printf("reader:DEBUG:Parse input\n");
	#endif
	
	int c;
	while (1)
	{
		int option_index = 0;
		static struct option long_options[] =
		{
			{  "blocks", required_argument, NULL, 'b' },
			{"boundary", optional_argument, NULL, 'B' },
			{   "count", required_argument, NULL, 'c' },
			{"datafile", required_argument, NULL, 'd' },
			{    "help",       no_argument, NULL, 'h' },
			{   "input", required_argument, NULL, 'i' },
			{ "ingroup", required_argument, NULL, 'I' },
			{ "logfile", required_argument, NULL, 'l' },
			{ "repeats", required_argument, NULL, 'r' },
			{ "verbose",       no_argument, NULL, 'v' },
			{"xthreads",       no_argument, NULL, 'x' },
			{"ythreads",       no_argument, NULL, 'y' },
			{         0,                 0,    0,   0 }
		};
		
		c = getopt_long(argc, argv, "b:B::c:d:hi:I:l:r:vx:y:", long_options, &option_index);
		
		if (c == -1)
		{
			break;
		}
		
		switch (c)
		{
			case 'b':
				#ifdef DEBUG
				printf("\tUsing ");
				if (optarg)
				{
					printf(" %d", atoi(optarg));
				}
				printf(" blocks per grid.\n");
				#endif
				blocks_per_grid = atoi(optarg);
				break;
			case 'B':
				#ifdef DEBUG
				printf("Staring on beginning of ");
				if (optarg)
				{
					printf("%d",atoi(optarg));
				}
				else
				{
					printf("0");
				}
				printf("th full B-engine frame.\n");
				#endif
				if (optarg)
				{
					beng_frame_offset = atoi(optarg);
				}
				else
				{
					beng_frame_offset = 1;
				}
				if (beng_frame_offset < 1)
				{
					fprintf(stderr,"B-engine frame offset should be positive, but given as %d.\n",beng_frame_offset);
					exit(EXIT_FAILURE);
				}
				break;
			case 'c':
				#ifdef DEBUG
				printf("\tReading");
				if (optarg)
				{
					printf(" %d", atoi(optarg));
				}
				printf(" VDIF frames.\n");
				#endif
				num_vdif_frames = atoi(optarg);
				break;
			case 'd':
				#ifdef DEBUG
				printf("\tDatafile is");
				if (optarg)
				{
					printf(" '%s'.", optarg);
				}
				printf("\n");
				#endif
				snprintf(filename_data, sizeof(filename_data), "%s", optarg);
				data_to_file = 1;
				break;
			case 'h':
				printf("Usage: %s [OPTIONS] -i <input_file>\n",argv[0]);
				printf("Options:\n");
				printf("  -b M, --blocks=M     Use <M> thread blocks for GPU kernel execution.\n");
				printf("  -B B, --boundary=B   Start reading VDIF packets offset by <B> B-engine frame counter values relative to the first encountered. <B> should be greater than 0.\n");
				printf("  -c N, --count=N      Read <N> VDIF frames from file <input_file>.\n");
				printf("  -d F, --datafile=F   Write B-engine data to <F>.\n");
				printf("  -i F, --input=F      Read data from file <F>.\n");
				printf("  -I F, --ingroup=F    Read data from group of files with prefix <F>.\n");
				printf("  -l F, --logfile=F    Activate logging to <F>.\n");
				printf("  -r R, --repeats=R    Repeat call to GPU kernel <R> times.\n");
				printf("  -v  , --verbose      Verbose output.\n");
				printf("\n");
				exit(EXIT_SUCCESS);
				break;
			case 'i':
				#ifdef DEBUG
				printf("\tInput file is");
				if (optarg)
				{
					printf(" '%s'.", optarg);
				}
				printf("\n");
				#endif
				snprintf(filename_input, sizeof(filename_input), "%s", optarg);
				break;
			case 'I':
				#ifdef DEBUG
				printf("\tInput group is");
				if (optarg)
				{
					printf(" '%s'.", optarg);
				}
				printf(" The following files will be read:\n");
				for (ii=2; ii<6; ii++)
				{
					printf("\t\t%s_eth%d.vdif\n",optarg,ii);
				}
				#endif
				snprintf(filename_input, sizeof(filename_input), "%s", optarg);
				input_is_batch = 1;
				break;
			case 'l':
				#ifdef DEBUG
				printf("\tLogfile is");
				if (optarg)
				{
					printf(" '%s'.", optarg);
				}
				printf("\n");
				#endif
				snprintf(filename_log, sizeof(filename_log), "%s", optarg);
				logging = 1;
				break;
			case 'r':
				#ifdef DEBUG
				printf("\tRunning ");
				if (optarg)
				{
					printf(" %d", atoi(optarg));
				}
				printf(" repeats.\n");
				#endif
				repeats = atoi(optarg);
				break;
			case 'v':
				#ifdef DEBUG
				printf("\tVerbose output.\n");
				#endif
				verbose_output = 1;
				break;
			case 'x':
				#ifdef DEBUG
				printf("\tUsing ");
				if (optarg)
				{
					printf(" %d", atoi(optarg));
				}
				printf(" x-threads.\n");
				#endif
				num_x_threads = atoi(optarg);
				break;
			case 'y':
				#ifdef DEBUG
				printf("\tUsing ");
				if (optarg)
				{
					printf(" %d", atoi(optarg));
				}
				printf(" y-threads.\n");
				#endif
				num_y_threads = atoi(optarg);
				break;
			default:
				fprintf(stderr,"?? getopt returned character code 0%o ??\n.",c);
				exit(EXIT_FAILURE);
		}
	}
	#ifdef DEBUG
	if (logging)
	{
		printf("reader:DEBUG:Opening file '%s' for logging.\n",filename_log);
	}
	#endif
	
	// open logfile
	if (logging)
	{
		if (strlen(filename_log) == 0)
		{
			fprintf(stderr,"Log filename not specified.\n");
			exit(EXIT_FAILURE);
		}
		else
		{
			fh_log = fopen(filename_log,"a");
			if (fh_log != NULL)
			{
				if (ftell(fh_log) == 0)
				{
					wall_clock = time(NULL);
					fprintf(fh_log,"#log-file created: %s",ctime(&wall_clock)); // ctime returns string with \n included
					fprintf(fh_log,"#    CPU [ms]    CUDA [ms]\n");
				}
			}
			else
			{
				fprintf(stderr,"Unable to open logfile '%s'.\n",filename_log);
				exit(EXIT_FAILURE);
			}
			fprintf(fh_log,"#Repeats: %d\n#Blocks per grid: %d\n#Threads-per-block (x,y): %d,%d\n",repeats,blocks_per_grid,num_x_threads,num_y_threads);
		}
	}
	
	#ifdef DEBUG
	if (data_to_file)
	{
		printf("reader:DEBUG:Opening file '%s' for data output.\n",filename_data);
	}
	#endif
	
	// open datafile
	if (data_to_file)
	{
		if (strlen(filename_data) == 0)
		{
			fprintf(stderr,"Data filename not specified.\n");
			exit(EXIT_FAILURE);
		}
		else
		{
			fh_data = fopen(filename_data,"w");
			if (fh_data != NULL)
			{
				// write the buffer size
				int32_t tmp = BENG_BUFFER_IN_COUNTS;
				fwrite((void *)&tmp, sizeof(int32_t), 1, fh_data);
			}
			else
			{
				fprintf(stderr,"Unable to open datafile '%s'.\n",filename_data);
				exit(EXIT_FAILURE);
			}
		}
	}
	
	#ifdef DEBUG
	printf("reader:DEBUG:Reading %d VDIF frames from '%s'.\n",num_vdif_frames,filename_input);
	#endif
	
	// read requested number of VDIF frames from input file
	if (num_vdif_frames < 1)
	{
		fprintf(stderr,"Number of frames has to be greater than 0 (given %d).\n",num_vdif_frames);
		exit(EXIT_FAILURE);
	}
	size_t num_vdif_bytes = num_vdif_frames*VDIF_BYTE_SIZE; // total bytes to read
	if (strlen(filename_input) == 0)
	{
		fprintf(stderr,"Input filename not specified.\n");
		exit(EXIT_FAILURE);
	}
	else
	{
		//~ vdif_buf = (int32_t *)malloc(num_vdif_bytes);
		err = hipHostAlloc((void **)&vdif_buf,num_vdif_bytes,hipHostMallocDefault);
		error_check(err,__FILE__,__LINE__);
		if (vdif_buf == NULL)
		{
			fprintf(stderr,"Unable to allocate memory for input data.\n");
			fclose(fh);
			exit(EXIT_FAILURE);
		}
		if (!input_is_batch)
		{
			fh = fopen(filename_input,"r");
			if (fh != NULL)
			{
				// if we start at some specified B-engine frame boundary
				fread((void *)tmp_vdif, VDIF_BYTE_SIZE, 1, fh);
				tmp_bcount_curr = get_bcount_from_vdif(tmp_vdif);
				if (beng_frame_offset > 0)
				{
					tmp_bcount_prev = tmp_bcount_curr;
					while (tmp_bcount_curr-tmp_bcount_prev < beng_frame_offset)
					{
						#ifdef DEBUG
							printf("reader:DEBUG:B-count = %d < %d, skipping.\n",tmp_bcount_curr,tmp_bcount_prev+beng_frame_offset);
						#endif
						fread((void *)tmp_vdif, VDIF_BYTE_SIZE, 1, fh);
						tmp_bcount_curr = get_bcount_from_vdif(tmp_vdif);
					}
					#ifdef DEBUG
						printf("reader:DEBUG:B-count = %d = %d, seeking one frame back.\n",tmp_bcount_curr,tmp_bcount_prev+beng_frame_offset);
					#endif
				}
				fseek(fh,-1*VDIF_BYTE_SIZE,SEEK_CUR);
				bcount_offset = tmp_bcount_curr;
				#ifdef DEBUG
				printf("reader:DEBUG:First B-engine counter value is %d.\n",bcount_offset);
				#endif
				// read file
				#ifdef DEBUG
				printf("reader:DEBUG:Reading %d x %d = %d bytes from file.\n",num_vdif_frames,num_vdif_bytes/num_vdif_frames,num_vdif_bytes);
				#endif
				size_t num_elem = fread((void *)vdif_buf, VDIF_BYTE_SIZE, num_vdif_frames, fh); 
				if (num_elem != num_vdif_frames)
				{
					fprintf(stderr,"Unable to read all the requested data.\n");
					fclose(fh);
					exit(EXIT_FAILURE);
				}
				fclose(fh);
			}
			else
			{
				fprintf(stderr,"Unable to open input file '%s'.\n",filename_input);
				exit(EXIT_FAILURE);
			}
		} // if (input_is_batch)...
		else
		{
			for (ii=2; ii<6; ii++)
			{
				char tmp_filename[0x100] = "\0";
				snprintf(tmp_filename, sizeof(tmp_filename), "%s_eth%d.vdif", filename_input,ii);
				#ifdef DEBUG
				printf("reader:DEBUG:Reading from %s in batch.\n",tmp_filename);
				#endif
				fh = fopen(tmp_filename,"r");
				if (fh != NULL)
				{
					// if we start at some specified B-engine frame boundary
					fread((void *)tmp_vdif, VDIF_BYTE_SIZE, 1, fh);
					tmp_bcount_curr = get_bcount_from_vdif(tmp_vdif);
					if (beng_frame_offset > 0)
					{
						tmp_bcount_prev = tmp_bcount_curr;
						while (tmp_bcount_curr-tmp_bcount_prev < beng_frame_offset)
						{
							#ifdef DEBUG
								printf("reader:DEBUG:B-count = %d < %d, skipping.\n",tmp_bcount_curr,tmp_bcount_prev+beng_frame_offset);
							#endif
							fread((void *)tmp_vdif, VDIF_BYTE_SIZE, 1, fh);
							tmp_bcount_curr = get_bcount_from_vdif(tmp_vdif);
						}
						#ifdef DEBUG
							printf("reader:DEBUG:B-count = %d = %d, seeking one frame back.\n",tmp_bcount_curr,tmp_bcount_prev+beng_frame_offset);
						#endif
					}
					fseek(fh,-1*VDIF_BYTE_SIZE,SEEK_CUR);
					bcount_offset = tmp_bcount_curr;
					#ifdef DEBUG
					printf("reader:DEBUG:First B-engine counter value is %d.\n",bcount_offset);
					#endif
					// read file
					#ifdef DEBUG
					printf("reader:DEBUG:Reading %d x %d = %d bytes from file.\n",num_vdif_frames/4,num_vdif_bytes/num_vdif_frames/4,num_vdif_bytes/4);
					printf("\tOffset is %d.\n",VDIF_INT_SIZE*num_vdif_frames/4);
					#endif
					size_t num_elem = fread((void *)(vdif_buf + (ii-2)*VDIF_INT_SIZE*num_vdif_frames/4), VDIF_BYTE_SIZE, num_vdif_frames/4, fh); 
					if (num_elem != num_vdif_frames/4)
					{
						fprintf(stderr,"Unable to read all the requested data.\n");
						fclose(fh);
						exit(EXIT_FAILURE);
					}
					fclose(fh);
				}
				else
				{
					fprintf(stderr,"Unable to open input file '%s'.\n",filename_input);
					exit(EXIT_FAILURE);
				}
			}
		} // if (input_is_batch) {} else ...
	}
	
	#ifdef DEBUG
	printf("reader:DEBUG:Creating CUDA events for timing.\n");
	#endif
	err = hipEventCreate(&start);
	error_check(err,__FILE__,__LINE__);
	err = hipEventCreate(&stop);
	error_check(err,__FILE__,__LINE__);
	
	#ifdef DEBUG
	printf("reader:DEBUG:Copying data from host to device...");
	#endif
	err = hipMalloc((void **)&gpu_vdif_buf, num_vdif_bytes);
	error_check(err,__FILE__,__LINE__);
	err = hipMemcpy(gpu_vdif_buf, vdif_buf, num_vdif_bytes, hipMemcpyHostToDevice);
	error_check(err,__FILE__,__LINE__);
	#ifdef DEBUG
	printf(" done.\n");
	#endif
	
	// define control structures
	#ifdef DEBUG
	printf("reader:DEBUG:Allocating memory for control...");
	#endif
	beng_frame_completion = (int32_t *)malloc(sizeof(int32_t)*BENG_BUFFER_IN_COUNTS);
	err = hipMalloc((void **)&gpu_beng_frame_completion, sizeof(int32_t)*BENG_BUFFER_IN_COUNTS);
	error_check(err,__FILE__,__LINE__);
	#ifdef DEBUG
	printf(" done.\n");
	#endif
	// initialize completion counter on host and device
	for (ii=0; ii<BENG_BUFFER_IN_COUNTS; ii++)
	{
		beng_frame_completion[ii] = 0;
	}
	#ifdef DEBUG
	printf("reader:DEBUG:Copying control from host to device...");
	#endif
	err = hipMemcpy(gpu_beng_frame_completion, beng_frame_completion, sizeof(int32_t)*BENG_BUFFER_IN_COUNTS, hipMemcpyHostToDevice);
	error_check(err,__FILE__,__LINE__);
	#ifdef DEBUG
	printf(" done.\n");
	#endif
	
	// output
	#ifdef DEBUG
	printf("reader:DEBUG:Allocating memory for output.\n");
	#endif
	cid = (int32_t *)malloc(num_vdif_frames*sizeof(int32_t));
	fid = (int32_t *)malloc(num_vdif_frames*sizeof(int32_t));
	bcount = (int32_t *)malloc(num_vdif_frames*sizeof(int32_t));
	
	size_t beng_data_bytes = BENG_CHANNELS*BENG_SNAPSHOTS*BENG_BUFFER_IN_COUNTS*sizeof(hipfftComplex);
	#ifdef DEBUG
	printf("reader:DEBUG:Allocating %d bytes for B-engine buffer.\n",2*beng_data_bytes);
	#endif
	//~ beng_data_0 = (cufftComplex *)malloc(beng_data_bytes);
	err = hipHostAlloc((void **)&beng_data_0,beng_data_bytes,hipHostMallocDefault);
	error_check(err,__FILE__,__LINE__);
	//~ beng_data_1 = (cufftComplex *)malloc(beng_data_bytes);
	err = hipHostAlloc((void **)&beng_data_1,beng_data_bytes,hipHostMallocDefault);
	error_check(err,__FILE__,__LINE__);
	
	// allocate memory for iFFT (host)
	size_t time_series_bytes = 2*BENG_CHANNELS_*BENG_SNAPSHOTS*BENG_BUFFER_IN_COUNTS*sizeof(hipfftReal);
	err = hipHostAlloc((void **)&time_series_0,time_series_bytes,hipHostMallocDefault);
	error_check(err,__FILE__,__LINE__);
	err = hipHostAlloc((void **)&time_series_1,time_series_bytes,hipHostMallocDefault);
	error_check(err,__FILE__,__LINE__);
	
	err = hipMalloc((void **)&gpu_cid, num_vdif_frames*sizeof(int32_t));
	error_check(err,__FILE__,__LINE__);
	err = hipMemset((void *)gpu_cid, 0, num_vdif_frames*sizeof(int32_t));
	error_check(err,__FILE__,__LINE__);
	err = hipMalloc((void **)&gpu_fid, num_vdif_frames*sizeof(int32_t));
	error_check(err,__FILE__,__LINE__);
	err = hipMemset((void *)gpu_fid, 0, num_vdif_frames*sizeof(int32_t));
	error_check(err,__FILE__,__LINE__);
	err = hipMalloc((void **)&gpu_bcount, num_vdif_frames*sizeof(int32_t));
	error_check(err,__FILE__,__LINE__);
	err = hipMemset((void *)gpu_bcount, 0, num_vdif_frames*sizeof(int32_t));
	error_check(err,__FILE__,__LINE__);
	err = hipMalloc((void **)&gpu_beng_data_0, beng_data_bytes);
	error_check(err,__FILE__,__LINE__);
	err = hipMemset((void *)gpu_beng_data_0, 0, beng_data_bytes);
	error_check(err,__FILE__,__LINE__);
	err = hipMalloc((void **)&gpu_beng_data_1, beng_data_bytes);
	error_check(err,__FILE__,__LINE__);
	err = hipMemset((void *)gpu_beng_data_1, 0, beng_data_bytes);
	error_check(err,__FILE__,__LINE__);
	hipDeviceSynchronize(); // make sure the memset is done
	
	// allocate memory for iFFT (device)
	err = hipMalloc((void **)&gpu_time_series_0, time_series_bytes);
	error_check(err,__FILE__,__LINE__);
	err = hipMemset((void *)gpu_time_series_0, 0, time_series_bytes);
	error_check(err,__FILE__,__LINE__);
	err = hipMalloc((void **)&gpu_time_series_1, time_series_bytes);
	error_check(err,__FILE__,__LINE__);
	err = hipMemset((void *)gpu_time_series_1, 0, time_series_bytes);
	error_check(err,__FILE__,__LINE__);
	
	// make iFFT plan
	#ifdef DEBUG
	printf("reader:DEBUG:Creating CUFFT plan. %d-dimensional %d-element iFFT in batch of %d.\n",ifft_rank,ifft_size,BENG_SNAPSHOTS*BENG_BUFFER_IN_COUNTS);
	#endif
	cures = hipfftPlanMany(&ifft_plan, ifft_rank, &ifft_size, 
			ifft_inembed, ifft_istride, ifft_idist,
			ifft_onembed, ifft_ostride, ifft_odist,
			HIPFFT_C2R, BENG_SNAPSHOTS*BENG_BUFFER_IN_COUNTS);
	if (cures != HIPFFT_SUCCESS)
	{
		fprintf(stderr,"CUFFT error:Could not create plan.\n");
		exit(EXIT_FAILURE);
	}
	
	#ifdef DEBUG
	printf("reader:DEBUG:Defining threads and blocks.\n");
	#endif
	dim3 threadsPerBlock(num_x_threads,num_y_threads);
	#ifdef DEBUG
	printf("\tthreads-per-block = (%d,%d,%d)\n",threadsPerBlock.x,threadsPerBlock.y,threadsPerBlock.z);
	#endif
	dim3 blocksPerGrid(blocks_per_grid);
	#ifdef DEBUG
	printf("\t  blocks-per-grid = (%d,%d,%d)\n",blocksPerGrid.x,blocksPerGrid.y,blocksPerGrid.z);
	#endif
	
	for (ir=0; ir<repeats; ir++)
	{
		#ifdef DEBUG
		printf("reader:DEBUG:Call to GPU kernel.\n");
		#endif
		hipEventRecord(start);
		hipEventSynchronize(start);
		clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&t0);
		vdif_to_beng<<<blocksPerGrid,threadsPerBlock>>>(gpu_vdif_buf,gpu_fid,gpu_cid,gpu_bcount,gpu_beng_data_0,gpu_beng_data_1,gpu_beng_frame_completion,num_vdif_frames,bcount_offset,blocks_per_grid);
		hipDeviceSynchronize();
		clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&t1);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_spent, start, stop);
		if (logging)
		{
			fprintf(fh_log,"   %10.6f",time_spent);
			fprintf(fh_log,"   %10.6f\n",1e3*(double)(t1.tv_sec - t0.tv_sec) + 1e-6*(double)(t1.tv_nsec - t0.tv_nsec));
		}
		else
		{
			printf("Reading VDIF frames finished in:\n\tCUDA: %10.6fms\n",time_spent);
			printf("\t CPU: %10.6fms\n",1e3*(double)(t1.tv_sec - t0.tv_sec) + 1e-6*(double)(t1.tv_nsec - t0.tv_nsec));
		}
	}
	
	#ifdef DEBUG
	printf("reader:DEBUG:Copying data from device to host...");
	#endif
	err = hipMemcpy(fid, gpu_fid, num_vdif_frames*sizeof(int32_t), hipMemcpyDeviceToHost);
	error_check(err,__FILE__,__LINE__);
	err = hipMemcpy(cid, gpu_cid, num_vdif_frames*sizeof(int32_t), hipMemcpyDeviceToHost);
	error_check(err,__FILE__,__LINE__);
	err = hipMemcpy(bcount, gpu_bcount, num_vdif_frames*sizeof(int32_t), hipMemcpyDeviceToHost);
	error_check(err,__FILE__,__LINE__);
	err = hipMemcpy(beng_data_0, gpu_beng_data_0, beng_data_bytes, hipMemcpyDeviceToHost);
	error_check(err,__FILE__,__LINE__);
	err = hipMemcpy(beng_data_1, gpu_beng_data_1, beng_data_bytes, hipMemcpyDeviceToHost);
	error_check(err,__FILE__,__LINE__);
	#ifdef DEBUG
	printf(" done.\n");
	#endif
	
	#ifdef DEBUG
	printf("reader:DEBUG:Copying control from device to host...");
	#endif
	err = hipMemcpy(beng_frame_completion, gpu_beng_frame_completion, sizeof(int32_t)*BENG_BUFFER_IN_COUNTS, hipMemcpyDeviceToHost);
	error_check(err,__FILE__,__LINE__);
	#ifdef DEBUG
	printf(" done.\n");
	#endif
	
	// iFFT
	for (ir=0; ir<repeats; ir++)
	{
		#ifdef DEBUG
		printf("reader:DEBUG:Call to GPU kernel.\n");
		#endif
		hipEventRecord(start);
		hipEventSynchronize(start);
		clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&t0);
		cures = hipfftExecC2R(ifft_plan, gpu_beng_data_0, gpu_time_series_0);
		hipDeviceSynchronize();
		if (cures != HIPFFT_SUCCESS)
		{
			fprintf(stderr,"CUFFT error:Could not execute iFFT on phased sum 0.\n");
			exit(EXIT_FAILURE);
		}
		cures = hipfftExecC2R(ifft_plan, gpu_beng_data_1, gpu_time_series_1);
		hipDeviceSynchronize();
		if (cures != HIPFFT_SUCCESS)
		{
			fprintf(stderr,"CUFFT error:Could not execute iFFT on phased sum 1.\n");
			exit(EXIT_FAILURE);
		}
		hipDeviceSynchronize();
		clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&t1);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_spent, start, stop);
		printf("iFFT finished in:\n\tCUDA: %10.6fms\n",time_spent);
		printf("\t CPU: %10.6fms\n",1e3*(double)(t1.tv_sec - t0.tv_sec) + 1e-6*(double)(t1.tv_nsec - t0.tv_nsec));
	}
	
	#ifdef DEBUG
	printf("reader:DEBUG:Copying data from device to host...");
	#endif
	err = hipMemcpy(time_series_0, gpu_time_series_0, time_series_bytes, hipMemcpyDeviceToHost);
	error_check(err,__FILE__,__LINE__);
	err = hipMemcpy(time_series_1, gpu_time_series_1, time_series_bytes, hipMemcpyDeviceToHost);
	error_check(err,__FILE__,__LINE__);
	#ifdef DEBUG
	printf(" done.\n");
	#endif
	
	if (data_to_file)
	{
		// write B-engine completion counters
		fwrite((void *)beng_frame_completion, sizeof(int32_t), BENG_BUFFER_IN_COUNTS, fh_data);
		// write B-engine data for phased sum 0
		fwrite((void *)beng_data_0, sizeof(hipfftComplex), BENG_CHANNELS*BENG_SNAPSHOTS*BENG_BUFFER_IN_COUNTS, fh_data);
		// write B-engine data for phased sum 1
		fwrite((void *)beng_data_1, sizeof(hipfftComplex), BENG_CHANNELS*BENG_SNAPSHOTS*BENG_BUFFER_IN_COUNTS, fh_data);
		// write time series data for phased sum 0
		fwrite((void *)time_series_0, sizeof(hipfftReal), 2*BENG_CHANNELS_*BENG_SNAPSHOTS*BENG_BUFFER_IN_COUNTS, fh_data);
		// write time series data for phased sum 1
		fwrite((void *)time_series_1, sizeof(hipfftReal), 2*BENG_CHANNELS_*BENG_SNAPSHOTS*BENG_BUFFER_IN_COUNTS, fh_data);
	}
	
	#ifdef DEBUG_SINGLE_FRAME
		for (ii=0; ii<num_vdif_frames; ii++)
		{
			if (cid[ii] == DEBUG_SINGLE_FRAME_CID && fid[ii] == DEBUG_SINGLE_FRAME_FID && bcount[ii] == DEBUG_SINGLE_FRAME_BCOUNT)
			{
				printf("B-count: %8d; fid: %3d; cid: %3d\n",bcount[ii],fid[ii],cid[ii]);
				int ch_a = SWARM_XENG_PARALLEL_CHAN * (cid[ii] * SWARM_N_FIDS + fid[ii]);
				printf("        a        b        c        d        e        f        g        h\n");
				for (ij=0; ij<SWARM_XENG_PARALLEL_CHAN; ij++)
				{
					printf("    %5d",ch_a+ij);
				}
				printf("\n");
				#ifdef BENG_FRAMES_OUT_CONSECUTIVE_SNAPSHOTS
					int idx_beng = ch_a*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS + ((bcount[ii]-bcount_offset)&BENG_BUFFER_INDEX_MASK)*BENG_SNAPSHOTS;
				#else
					int idx_beng = BENG_CHANNELS*BENG_SNAPSHOTS*((bcount[ii]-bcount_offset)&BENG_BUFFER_INDEX_MASK) + ch_a;
				#endif
				for (ij=0; ij<BENG_SNAPSHOTS; ij++)
				{
					printf("\tSnapshot #%3d:\n",ij);
					for (ik=0; ik<SWARM_XENG_PARALLEL_CHAN; ik++)
					{
						#ifdef BENG_FRAMES_OUT_CONSECUTIVE_SNAPSHOTS
							printf("   %2d%+2dj",(int)cuCrealf(beng_data_0[idx_beng+ij+BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS*ik]),(int)cuCimagf(beng_data_0[idx_beng+ij+BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS*ik]));
						#else
							printf("   %2d%+2dj",(int)cuCrealf(beng_data_0[idx_beng+BENG_CHANNELS*ij+ik]),(int)cuCimagf(beng_data_0[idx_beng+BENG_CHANNELS*ij+ik]));
						#endif
						
					}
					printf("\n");
					for (ik=0; ik<SWARM_XENG_PARALLEL_CHAN; ik++)
					{
						#ifdef BENG_FRAMES_OUT_CONSECUTIVE_SNAPSHOTS
							printf("   %2d%+2dj",(int)cuCrealf(beng_data_1[idx_beng+ij+BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS*ik]),(int)cuCimagf(beng_data_1[idx_beng+ij+BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS*ik]));
						#else
							printf("   %2d%+2dj",(int)cuCrealf(beng_data_1[idx_beng+BENG_CHANNELS*ij+ik]),(int)cuCimagf(beng_data_1[idx_beng+BENG_CHANNELS*ij+ik]));
						#endif
					}
					printf("\n");
				}
			}
		}
	#endif
	
	// Free device global memory
	#ifdef DEBUG
	printf("reader:DEBUG:Free device memory.\n");
	#endif
	hipFree(gpu_vdif_buf);
	hipFree(gpu_cid);
	hipFree(gpu_fid);
	hipFree(gpu_bcount);
	hipFree(gpu_beng_data_0);
	hipFree(gpu_beng_data_1);
	hipFree(gpu_beng_frame_completion);
	// Free host memory
	#ifdef DEBUG
	printf("reader:DEBUG:Free host memory.\n");
	#endif
	//~ free(vdif_buf);
	hipHostFree((void *)vdif_buf);
	free(cid);
	free(fid);
	free(bcount);
	//~ free(beng_data_0);
	hipHostFree((void *)beng_data_0);
	//~ free(beng_data_1);
	hipHostFree((void *)beng_data_1);
	free(beng_frame_completion);
	
	// Destroy cufft plan and free memory
	hipfftDestroy(ifft_plan);
	hipHostFree(time_series_0);
	hipHostFree(time_series_1);
	hipFree(gpu_time_series_0);
	hipFree(gpu_time_series_1);
	
	// Reset the device and exit
	#ifdef DEBUG
	printf("reader:DEBUG:CUDA Device reset.\n");
	#endif
	hipDeviceReset();
	
	#ifdef DEBUG
	printf("reader:DEBUG:Stop.\n");
	#endif
	
	if (logging)
	{
		fclose(fh_log);
	}
	if (data_to_file)
	{
		fclose(fh_data);
	}
	exit(EXIT_SUCCESS);
}

/*
 * Parse VDIF frame and store B-engine frames in buffer.
 * */
__global__ void vdif_to_beng(
	int32_t *vdif_frames, 
	int32_t *fid_out, 
	int32_t *cid_out, 
	int32_t *bcount_out, 
	hipfftComplex *beng_data_out_0, 
	hipfftComplex *beng_data_out_1, 
	int32_t *beng_frame_completion,
	int32_t num_vdif_frames, 
	int32_t bcount_offset,
	int blocks_per_grid)
{
	// VDIF header
	int32_t cid,fid;
	int32_t bcount; // we don't need very large bcount, just keep lower 32bits
	
	// VDIF data
	const int32_t *vdif_frame_start; // pointer to start of the VDIF frame handled by this thread
	int32_t samples_per_snapshot_half_0, samples_per_snapshot_half_1; // 4byte collections of 16 2bit samples (2sums * (1real + 1imag) * 4xeng_parallel_chan) each
	int32_t idx_beng_data_out; // index into beng_data_out
	
	#ifdef GPUCTRL_SHARED_RAW_VDIF
		__shared__ int32_t vdif_shared[blockDim.y][VDIF_INT_SIZE];
	#endif
	
	// misc
	int32_t iframe; // VDIF frame sized index into VDIF data buffer currently processed by this thread
	int idata; // uint64_t sized index into VDIF data currently processed by this thread
	int isample; // 2bit sized index into the 8 consecutive channels of a single snapshot contained in each B-engine packet
	
	// control
	int old; // old value for B-engine completion counter
	
	/* iframe increases by the number of frames handled by a single grid.
	 * There are blocks_per_grid*blockDim.y frames handled simultaneously
	 * withing the grid.
	 * */
	for (iframe=0; iframe + threadIdx.y + blockIdx.x*blockDim.y<num_vdif_frames; iframe+=blocks_per_grid*blockDim.y)
	{ 
		
		#ifdef DEBUG_GPU
			#ifdef DEBUG_SINGLE_FRAME
				if (cid == DEBUG_SINGLE_FRAME_CID && fid == DEBUG_SINGLE_FRAME_FID && bcount == DEBUG_SINGLE_FRAME_BCOUNT)
				{
			#endif // DEBUG_SINGLE_FRAME
			#ifdef DEBUG_GPU_CONDITION
					if ( DEBUG_GPU_CONDITION )
					{
			#endif // DEBUG_GPU_CONDITION
						printf("blk(thx,thy)=%3d(%3d,%3d): #frame = %d + %d + %d*%d = %d < %d ? %s\n",blockIdx.x,threadIdx.x,threadIdx.y,
						iframe , threadIdx.y , blockIdx.x , blockDim.y,
						iframe + threadIdx.y + blockIdx.x*blockDim.y,num_vdif_frames,
						iframe + threadIdx.y + blockIdx.x*blockDim.y < num_vdif_frames ? "OK" : "NO");
			#ifdef DEBUG_GPU_CONDITION
					}
			#endif // DEBUG_GPU_CONDITION
			#ifdef DEBUG_SINGLE_FRAME
				}
			#endif // DEBUG_SINGLE_FRAME
		#endif // DEBUG_GPU
		
		/* Set the start of the VDIF frame handled by this thread. VDIF 
		 * frames are just linearly packed in memory. Consecutive y-threads
		 * read consecutive VDIF frames, and each x-block reads consecutive
		 * blocks of blockDim.y VDIF frames.
		 * */
		vdif_frame_start = vdif_frames + (iframe + threadIdx.y + blockIdx.x*blockDim.y)*VDIF_INT_SIZE;
		
		#ifdef GPUCTRL_SHARED_RAW_VDIF
			for (idata=threadIdx.x; idata<VDIF_INT_SIZE; idata+=blockDim.x)
			{
				vdif_shared[threadIdx.y][idata] = *(vdif_frame_start + idata);
			}
			__syncthreads();
		#endif
		
		#ifdef GPUCTRL_SHARED_RAW_VDIF
			cid = get_cid_from_vdif(vdif_shared[threadIdx.y]);
			fid = get_fid_from_vdif(vdif_shared[threadIdx.y]);
			bcount = get_bcount_from_vdif(vdif_shared[threadIdx.y]);
		#else
			cid = get_cid_from_vdif(vdif_frame_start);
			fid = get_fid_from_vdif(vdif_frame_start);
			bcount = get_bcount_from_vdif(vdif_frame_start);
		#endif
		
		cid_out[iframe + threadIdx.y + blockIdx.x*blockDim.y] = cid;
		fid_out[iframe + threadIdx.y + blockIdx.x*blockDim.y] = fid;
		bcount_out[iframe + threadIdx.y + blockIdx.x*blockDim.y] = bcount;
		
		#ifdef DEBUG_SINGLE_FRAME
			if (cid == DEBUG_SINGLE_FRAME_CID && fid == DEBUG_SINGLE_FRAME_FID && bcount == DEBUG_SINGLE_FRAME_BCOUNT)
			{
				// do nothing
			}
			else
			{
				continue;
			}
		#endif
		
		#ifdef BENG_FRAMES_OUT_CONSECUTIVE_SNAPSHOTS
			/* Reorder to have snapshots contiguous and consecutive channels
			 * separated by 128 snapshots times the number of B-engine frames
			 * in buffer.
			 */
			idx_beng_data_out = SWARM_XENG_PARALLEL_CHAN * (cid * SWARM_N_FIDS + fid)*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS;
			idx_beng_data_out += ((bcount-bcount_offset)&BENG_BUFFER_INDEX_MASK)*BENG_SNAPSHOTS;
		#else
			/* Set the offset into the B-engine data buffer. Channels for 
			 * a single snapshot are consecutive in memory, consecutive 
			 * snapshots are separated by one spectrum, and consecutive
			 * B-engine frames are separated by 128 snapshots (128 spectra).
			 * */
			idx_beng_data_out  = BENG_CHANNELS*BENG_SNAPSHOTS*((bcount-bcount_offset)&BENG_BUFFER_INDEX_MASK); // offset given the masked B-engine counter value
			idx_beng_data_out += SWARM_XENG_PARALLEL_CHAN * (cid * SWARM_N_FIDS + fid); // offset given the cid and fid
		#endif
		
		#ifdef BENG_FRAMES_OUT_CONSECUTIVE_SNAPSHOTS
			/* Reorder to have snapshots contiguous and consecutive channels
			 * separated by 128 snapshots times the number of B-engine frames
			 * in buffer. This means consecutive threads along x will handle
			 * consecutive snapshots.
			 */
			idx_beng_data_out += threadIdx.x;
		#else
			/* Add offset based on the threadIdx.x. Consecutive x-threads
			 * read consecutive 2-int32_t (8byte) data chunks, which means
			 * that the target index for consecutive x-threads are separated
			 * as consecutive snapshots, i.e. single spectrum.
			 * */
			idx_beng_data_out += threadIdx.x*BENG_CHANNELS; // offset given the threadIdx.x
		#endif
		
		#ifdef DEBUG_GPU
			#ifdef DEBUG_SINGLE_FRAME
				if (cid == DEBUG_SINGLE_FRAME_CID && fid == DEBUG_SINGLE_FRAME_FID && bcount == DEBUG_SINGLE_FRAME_BCOUNT)
				{
			#endif // DEBUG_SINGLE_FRAME
			#ifdef DEBUG_GPU_CONDITION
					if ( DEBUG_GPU_CONDITION )
					{
			#endif // DEBUG_GPU_CONDITION
						#ifdef BENG_FRAMES_OUT_CONSECUTIVE_SNAPSHOTS
							printf("blk(thx,thy)=%3d(%3d,%3d): cid=%3d, fid=%d, bcount=%8d (masked=%3d); (before data loop) idx_beng_data_out = %d*(%3d*%d+%d)*%d*%d + %d*%d + %d =  %9d + %9d + %9d = %9d --> %9d.\n",
									blockIdx.x,threadIdx.x,threadIdx.y,cid,fid,bcount,((bcount-bcount_offset)&BENG_BUFFER_INDEX_MASK),
									SWARM_XENG_PARALLEL_CHAN , cid , SWARM_N_FIDS , fid,BENG_BUFFER_IN_COUNTS,BENG_SNAPSHOTS,
									((bcount-bcount_offset)&BENG_BUFFER_INDEX_MASK),BENG_SNAPSHOTS,
									threadIdx.x,
									SWARM_XENG_PARALLEL_CHAN * (cid * SWARM_N_FIDS + fid)*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS,
									((bcount-bcount_offset)&BENG_BUFFER_INDEX_MASK)*BENG_SNAPSHOTS,
									threadIdx.x,
									SWARM_XENG_PARALLEL_CHAN * (cid * SWARM_N_FIDS + fid)*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS+((bcount-bcount_offset)&BENG_BUFFER_INDEX_MASK)*BENG_SNAPSHOTS+threadIdx.x,
									idx_beng_data_out);
						#else
							printf("blk(thx,thy)=%3d(%3d,%3d): cid=%3d, fid=%d, bcount=%8d (masked=%3d); (before data loop) idx_beng_data_out = %d*%d*%d + %d*(%3d*%d+%d) + %d*%d =  %9d + %9d + %9d = %9d --> %9d.\n",
									blockIdx.x,threadIdx.x,threadIdx.y,cid,fid,bcount,bcount & BENG_BUFFER_INDEX_MASK,
									BENG_CHANNELS,BENG_SNAPSHOTS,(bcount & BENG_BUFFER_INDEX_MASK),
									SWARM_XENG_PARALLEL_CHAN , cid , SWARM_N_FIDS , fid,
									threadIdx.x,BENG_CHANNELS,
									BENG_CHANNELS*BENG_SNAPSHOTS*(bcount & BENG_BUFFER_INDEX_MASK),
									SWARM_XENG_PARALLEL_CHAN * (cid * SWARM_N_FIDS + fid),
									threadIdx.x*BENG_CHANNELS,
									BENG_CHANNELS*BENG_SNAPSHOTS*(bcount & BENG_BUFFER_INDEX_MASK)+SWARM_XENG_PARALLEL_CHAN * (cid * SWARM_N_FIDS + fid)+threadIdx.x*BENG_CHANNELS,
									idx_beng_data_out);
						#endif
			#ifdef DEBUG_GPU_CONDITION
					}
			#endif // DEBUG_GPU_CONDITION
			#ifdef DEBUG_SINGLE_FRAME
				}
			#endif // DEBUG_SINGLE_FRAME
		#endif // DEBUG_GPU
		
		/* idata increases by the number of int32_t handled simultaneously
		 * by all x-threads. Each thread handles B-engine packet data 
		 * for a single snapshot per iteration.
		 * */
		for (idata=0; idata<VDIF_INT_SIZE_DATA; idata+=BENG_VDIF_INT_PER_SNAPSHOT*blockDim.x)
		{
			/* Get sample data out of global memory. Offset from the 
			 * VDIF frame start by the header, the number of snapshots
			 * processed by the group of x-threads (idata), and the
			 * particular snapshot offset for THIS x-thread 
			 * (BENG_VDIF_INT_PER_SNAPSHOT*threadIdx.x).
			 * */
			#ifdef GPUCTRL_SHARED_RAW_VDIF
				samples_per_snapshot_half_0 = *(vdif_shared[threadIdx.y] + VDIF_INT_SIZE_HEADER + idata + BENG_VDIF_INT_PER_SNAPSHOT*threadIdx.x);
				samples_per_snapshot_half_1 = *(vdif_shared[threadIdx.y] + VDIF_INT_SIZE_HEADER + idata + BENG_VDIF_INT_PER_SNAPSHOT*threadIdx.x + 1);
			#else
				samples_per_snapshot_half_0 = *(vdif_frame_start + VDIF_INT_SIZE_HEADER + idata + BENG_VDIF_INT_PER_SNAPSHOT*threadIdx.x);
				samples_per_snapshot_half_1 = *(vdif_frame_start + VDIF_INT_SIZE_HEADER + idata + BENG_VDIF_INT_PER_SNAPSHOT*threadIdx.x + 1);
			#endif
			for (isample=0; isample<SWARM_XENG_PARALLEL_CHAN/2; isample++)
			{
				#ifdef DEBUG_SINGLE_FRAME
					int32_t tmp_s0 = samples_per_snapshot_half_0;
					int32_t tmp_s1 = samples_per_snapshot_half_1;
				#endif
				
				#ifdef BENG_FRAMES_OUT_CONSECUTIVE_SNAPSHOTS
					beng_data_out_1[idx_beng_data_out+(SWARM_XENG_PARALLEL_CHAN/2-(isample+1))*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS] = read_complex_sample(&samples_per_snapshot_half_0);
					beng_data_out_0[idx_beng_data_out+(SWARM_XENG_PARALLEL_CHAN/2-(isample+1))*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS] = read_complex_sample(&samples_per_snapshot_half_0);
					beng_data_out_1[idx_beng_data_out+(SWARM_XENG_PARALLEL_CHAN/2-(isample+1)+SWARM_XENG_PARALLEL_CHAN/2)*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS] = read_complex_sample(&samples_per_snapshot_half_1);
					beng_data_out_0[idx_beng_data_out+(SWARM_XENG_PARALLEL_CHAN/2-(isample+1)+SWARM_XENG_PARALLEL_CHAN/2)*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS] = read_complex_sample(&samples_per_snapshot_half_1);
				#else
					beng_data_out_1[idx_beng_data_out+SWARM_XENG_PARALLEL_CHAN/2-(isample+1)] = read_complex_sample(&samples_per_snapshot_half_0);
					beng_data_out_0[idx_beng_data_out+SWARM_XENG_PARALLEL_CHAN/2-(isample+1)] = read_complex_sample(&samples_per_snapshot_half_0);
					beng_data_out_1[idx_beng_data_out+SWARM_XENG_PARALLEL_CHAN/2-(isample+1)+SWARM_XENG_PARALLEL_CHAN/2] = read_complex_sample(&samples_per_snapshot_half_1);
					beng_data_out_0[idx_beng_data_out+SWARM_XENG_PARALLEL_CHAN/2-(isample+1)+SWARM_XENG_PARALLEL_CHAN/2] = read_complex_sample(&samples_per_snapshot_half_1);
				#endif
				
				#ifdef DEBUG_SINGLE_FRAME
					int r1,r2,r3,r4,i1,i2,i3,i4;
					#ifdef BENG_FRAMES_OUT_CONSECUTIVE_SNAPSHOTS
						r1 = (int)(beng_data_out_1[idx_beng_data_out+(SWARM_XENG_PARALLEL_CHAN/2-(isample+1))*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS].x);
						i1 = (int)(beng_data_out_1[idx_beng_data_out+(SWARM_XENG_PARALLEL_CHAN/2-(isample+1))*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS].y);
						r2 = (int)(beng_data_out_0[idx_beng_data_out+(SWARM_XENG_PARALLEL_CHAN/2-(isample+1))*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS].x);
						i2 = (int)(beng_data_out_0[idx_beng_data_out+(SWARM_XENG_PARALLEL_CHAN/2-(isample+1))*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS].y);
						r3 = (int)(beng_data_out_1[idx_beng_data_out+(SWARM_XENG_PARALLEL_CHAN/2-(isample+1)+SWARM_XENG_PARALLEL_CHAN/2)*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS].x);
						i3 = (int)(beng_data_out_1[idx_beng_data_out+(SWARM_XENG_PARALLEL_CHAN/2-(isample+1)+SWARM_XENG_PARALLEL_CHAN/2)*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS].y);
						r4 = (int)(beng_data_out_0[idx_beng_data_out+(SWARM_XENG_PARALLEL_CHAN/2-(isample+1)+SWARM_XENG_PARALLEL_CHAN/2)*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS].x);
						i4 = (int)(beng_data_out_0[idx_beng_data_out+(SWARM_XENG_PARALLEL_CHAN/2-(isample+1)+SWARM_XENG_PARALLEL_CHAN/2)*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS].y);
					#else
						r1 = (int)(beng_data_out_1[idx_beng_data_out+SWARM_XENG_PARALLEL_CHAN/2-(isample+1)].x);
						i1 = (int)(beng_data_out_1[idx_beng_data_out+SWARM_XENG_PARALLEL_CHAN/2-(isample+1)].y);
						r2 = (int)(beng_data_out_0[idx_beng_data_out+SWARM_XENG_PARALLEL_CHAN/2-(isample+1)].x);
						i2 = (int)(beng_data_out_0[idx_beng_data_out+SWARM_XENG_PARALLEL_CHAN/2-(isample+1)].y);
						r3 = (int)(beng_data_out_1[idx_beng_data_out+SWARM_XENG_PARALLEL_CHAN/2-(isample+1)+SWARM_XENG_PARALLEL_CHAN/2].x);
						i3 = (int)(beng_data_out_1[idx_beng_data_out+SWARM_XENG_PARALLEL_CHAN/2-(isample+1)+SWARM_XENG_PARALLEL_CHAN/2].y);
						r4 = (int)(beng_data_out_0[idx_beng_data_out+SWARM_XENG_PARALLEL_CHAN/2-(isample+1)+SWARM_XENG_PARALLEL_CHAN/2].x);
						i4 = (int)(beng_data_out_0[idx_beng_data_out+SWARM_XENG_PARALLEL_CHAN/2-(isample+1)+SWARM_XENG_PARALLEL_CHAN/2].y);
					#endif
					
					if (cid == DEBUG_SINGLE_FRAME_CID && fid == DEBUG_SINGLE_FRAME_FID && bcount == DEBUG_SINGLE_FRAME_BCOUNT)
					{
						#ifdef DEBUG_GPU
							#ifdef DEBUG_SINGLE_FRAME
								if (cid == DEBUG_SINGLE_FRAME_CID && fid == DEBUG_SINGLE_FRAME_FID && bcount == DEBUG_SINGLE_FRAME_BCOUNT)
								{
							#endif // DEBUG_SINGLE_FRAME
							#ifdef DEBUG_GPU_CONDITION
									if ( DEBUG_GPU_CONDITION )
									{
							#endif // DEBUG_GPU_CONDITION
										printf("blk(thx,thy)=%3d(%3d,%3d): cid=%3d, fid=%d, bcount=%8d (masked=%3d); 0x%08x: 0x%02x = %3u -> (%2d,%2d) (%2d,%2d) ; 0x%08x: 0x%02x = %3u -> (%2d,%2d) (%2d,%2d) \n",
												blockIdx.x,threadIdx.x,threadIdx.y,cid,fid,bcount,(bcount-bcount_offset)&BENG_BUFFER_INDEX_MASK,
												tmp_s0,tmp_s0&0xFF,tmp_s0&0xFF,r1,i1,r2,i2,
												tmp_s1,tmp_s1&0xFF,tmp_s1&0xFF,r3,i3,r4,i4);
							#ifdef DEBUG_GPU_CONDITION
									}
							#endif // DEBUG_GPU_CONDITION
							#ifdef DEBUG_SINGLE_FRAME
								}
							#endif // DEBUG_SINGLE_FRAME
						#endif // DEBUG_GPU
					} // DEBUG_SINGLE_FRAME condition
				#endif // DEBUG_SINGLE_FRAME
				
			} // for (isample=0; ...)
			#ifdef BENG_FRAMES_OUT_CONSECUTIVE_SNAPSHOTS
				/* The next snapshot handled by this thread will increment
				 * by the number of x-threads, so index into B-engine data
				 * should increment by that number.
				 */
				idx_beng_data_out += blockDim.x;
			#else
				/* The next snapshot handled by this thread will increment
				 * by the number of x-threads, so index into B-engine data
				 * should increment by that many spectra.
				 * */
				idx_beng_data_out += blockDim.x*BENG_CHANNELS;
			#endif
		} // for (idata=0; ...)
		
		#ifdef DEBUG_GPU
			#ifdef DEBUG_SINGLE_FRAME
				if (cid == DEBUG_SINGLE_FRAME_CID && fid == DEBUG_SINGLE_FRAME_FID && bcount == DEBUG_SINGLE_FRAME_BCOUNT)
				{
			#endif // DEBUG_SINGLE_FRAME
			#ifdef DEBUG_GPU_CONDITION
					if ( DEBUG_GPU_CONDITION )
					{
			#endif // DEBUG_GPU_CONDITION
						#ifdef BENG_FRAMES_OUT_CONSECUTIVE_SNAPSHOTS
							printf("blk(thx,thy)=%3d(%3d,%3d): cid=%3d, fid=%d, bcount=%8d (masked=%3d); (after data loop) idx_beng_data_out = %d*(%3d*%d+%d)*%d*%d + %d*%d + %d =  %9d + %9d + %9d = %9d --> %9d.\n",
									blockIdx.x,threadIdx.x,threadIdx.y,cid,fid,bcount,((bcount-bcount_offset)&BENG_BUFFER_INDEX_MASK),
									SWARM_XENG_PARALLEL_CHAN , cid , SWARM_N_FIDS , fid,BENG_BUFFER_IN_COUNTS,BENG_SNAPSHOTS,
									((bcount-bcount_offset)&BENG_BUFFER_INDEX_MASK),BENG_SNAPSHOTS,
									threadIdx.x,
									SWARM_XENG_PARALLEL_CHAN * (cid * SWARM_N_FIDS + fid)*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS,
									((bcount-bcount_offset)&BENG_BUFFER_INDEX_MASK)*BENG_SNAPSHOTS,
									threadIdx.x,
									SWARM_XENG_PARALLEL_CHAN * (cid * SWARM_N_FIDS + fid)*BENG_BUFFER_IN_COUNTS*BENG_SNAPSHOTS+((bcount-bcount_offset)&BENG_BUFFER_INDEX_MASK)*BENG_SNAPSHOTS+threadIdx.x,
									idx_beng_data_out);
						#else
							printf("blk(thx,thy)=%3d(%3d,%3d): cid=%3d, fid=%d, bcount=%8d (masked=%3d); (after data loop) idx_beng_data_out = %d*%d*%d + %d*(%3d*%d+%d) + %d*%d =  %9d + %9d + %9d = %9d --> %9d.\n",
									blockIdx.x,threadIdx.x,threadIdx.y,cid,fid,bcount,bcount & BENG_BUFFER_INDEX_MASK,
									BENG_CHANNELS,BENG_SNAPSHOTS,(bcount & BENG_BUFFER_INDEX_MASK),
									SWARM_XENG_PARALLEL_CHAN , cid , SWARM_N_FIDS , fid,
									threadIdx.x,BENG_CHANNELS,
									BENG_CHANNELS*BENG_SNAPSHOTS*(bcount & BENG_BUFFER_INDEX_MASK),
									SWARM_XENG_PARALLEL_CHAN * (cid * SWARM_N_FIDS + fid),
									threadIdx.x*BENG_CHANNELS,
									BENG_CHANNELS*BENG_SNAPSHOTS*(bcount & BENG_BUFFER_INDEX_MASK)+SWARM_XENG_PARALLEL_CHAN * (cid * SWARM_N_FIDS + fid)+threadIdx.x*BENG_CHANNELS,
									idx_beng_data_out);
						#endif
			#ifdef DEBUG_GPU_CONDITION
					}
			#endif // DEBUG_GPU_CONDITION
			#ifdef DEBUG_SINGLE_FRAME
				}
			#endif // DEBUG_SINGLE_FRAME
		#endif // DEBUG_GPU
		
		//~ // TODO: reset completion counter for two B-engine frames behind, something like:
		//~ beng_frame_completion[(bcount-bcount_offset+BENG_BUFFER_IN_COUNTS-3)&BENG_BUFFER_INDEX_MASK] = 0;
		
		// increment completion counter for this B-engine frame
		old = atomicAdd(beng_frame_completion + ((bcount-bcount_offset)&BENG_BUFFER_INDEX_MASK), 1);
		#ifdef DEBUG_GPU
			#ifdef DEBUG_SINGLE_FRAME
				if (cid == DEBUG_SINGLE_FRAME_CID && fid == DEBUG_SINGLE_FRAME_FID && bcount == DEBUG_SINGLE_FRAME_BCOUNT)
				{
			#endif // DEBUG_SINGLE_FRAME
			#ifdef DEBUG_GPU_CONDITION
					if ( DEBUG_GPU_CONDITION )
					{
			#endif // DEBUG_GPU_CONDITION
						printf("blk(thx,thy)=%d(%d,%d): B-engine frame bcount=%8d (masked=%3d) completion increment: %6d --> %6d (FULL = %6d).\n",
								blockIdx.x,threadIdx.x,threadIdx.y,bcount,((bcount-bcount_offset)&BENG_BUFFER_INDEX_MASK),old,old+1,BENG_FRAME_COMPLETION_COMPLETE_ON_GPU);
			#ifdef DEBUG_GPU_CONDITION
					}
			#endif // DEBUG_GPU_CONDITION
			#ifdef DEBUG_SINGLE_FRAME
				}
			#endif // DEBUG_SINGLE_FRAME
		#endif // DEBUG_GPU
		
		/* Vote to see if the frame is complete. This will be indicated
		 * by the old value of the counter being one less than what indicates
		 * a full frame in one of the threads.
		 * */
		if (__any(old == BENG_FRAME_COMPLETION_COMPLETE_ON_GPU-1))
		{
			// do something...
			#ifdef DEBUG_GPU
				#ifdef DEBUG_SINGLE_FRAME
					if (cid == DEBUG_SINGLE_FRAME_CID && fid == DEBUG_SINGLE_FRAME_FID && bcount == DEBUG_SINGLE_FRAME_BCOUNT)
					{
				#endif // DEBUG_SINGLE_FRAME
				#ifdef DEBUG_GPU_CONDITION
						if ( DEBUG_GPU_CONDITION )
						{
				#endif // DEBUG_GPU_CONDITION
							printf("blk(thx,thy)=%d(%d,%d): B-engine frame bcount=%8d (masked=%3d) complete.\n",
									blockIdx.x,threadIdx.x,threadIdx.y,bcount,((bcount-bcount_offset)&BENG_BUFFER_INDEX_MASK));
				#ifdef DEBUG_GPU_CONDITION
						}
				#endif // DEBUG_GPU_CONDITION
				#ifdef DEBUG_SINGLE_FRAME
					}
				#endif // DEBUG_SINGLE_FRAME
			#endif // DEBUG_GPU
		}
	} // for (iframe=0; ...)
}

/*
 * CUDA error code checker.
 * 
 * Tests whether the CUDA error code returned is an error or success. In
 * case of error a message is displayed and the program exits.
 */
inline void error_check(const char *f, const int l)
{
	hipError_t err = hipGetLastError();
	error_check(err, f, l);
}
inline void error_check(hipError_t err, const char *f, const int l)
{
	if (err != hipSuccess)
	{
		fprintf(stderr, "CUDA error:%s.%d: %s\n", f, l, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
